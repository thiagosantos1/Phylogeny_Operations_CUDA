#include "hip/hip_runtime.h"
/*************************************************************************
	
	Copyright (C) 2016	Evandro Taquary, Thiago Santos
	
	This program is free software: you can redistribute it and/or modify s
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with this program.  If not, see <http://www.gnu.org/licenses/>.
	
*************************************************************************/

#include <iostream>
#include <string>
#include <fstream>
#include <hiprand/hiprand_kernel.h>
#include "modcpy.h"
#include <regex>
#include <unordered_map>
#include <iomanip>
#include <sys/time.h>

using namespace std;

#define START_TIMER() \
		{ \
			gettimeofday(&tv, NULL); \
			start_time = tv.tv_sec * 1000000 + tv.tv_usec; \
		}
//return time measurement in s
#define STOP_TIMER(time_spent) \
		{ \
			gettimeofday(&tv, NULL); \
			end_time = tv.tv_sec * 1000000 + tv.tv_usec; \
			time_spent = ((double)(end_time-start_time))/1000000; \
		}
#define CHECK(call) \
		{ \
			const hipError_t error = call; \
			if (error != hipSuccess) { \
				cout << "Error: " << __FILE__ ": " << __LINE__ << ", "; \
				cout << "code: "<< error << ", reason: " << hipGetErrorString(error) << endl; \
				exit(EXIT_FAILURE); \
			} \
		}

#define FERR(file) \
		{ \
			if(!file.good()){ \
				cout << "Something went wrong while reading the file! Please try again." << endl; \
				cout << "Error: " << __FILE__ ": " << __LINE__ << ", " << endl; \
				exit(EXIT_FAILURE); \
			} \
		}

typedef struct {
	ushort side	: 1;
	ushort idx	: 15;
} paren_t;

#define NOCHILD USHRT_MAX		//16 bits
#define NOPARENT USHRT_MAX/2	//15 bits

class SoaTree {
	private:
		paren_t	*parent;	// nodes' parents or the subtrees' roots' indices where new nodes shall be inserted (MDCC)
		ushort	*lChild;	// nodes' left children
		ushort	*rChild;	// nodes' right children
		float	*branch;	// lengths of the nodes' branches (distance to the parent)
		float	*dRoot;		// distances between nodes and root (sum of the paths' branches)
		ushort	*inseq;		// vector with the sequence of indices of puts to be inserted
		float	*vTraits;	// keep the value of the traits from all elements of the tree

	public:
		__host__ SoaTree() = default;
		__host__ SoaTree(int num_nodes, int num_ins) {soalloc(num_nodes,num_ins);}
		__host__ SoaTree(int num_nodes, void* base) {setOffs(num_nodes,base);}
		__host__ __device__ void* getPtr() const {return (void*) parent;}
		__host__ __device__ void setOffs(int num_nodes, void* base);	//set pointers' offsets starting on base accordingly to data structure, # nodes and # insertions
		__host__ void setOffs(int num_nodes) {setOffs(num_nodes, parent);}  //set pointers' offsets starting on the first array accordingly to data structure, # nodes and # insertions
		__host__ static size_t getSize(int num_nodes, int num_ins)
		{
			size_t size = (3*sizeof(ushort) + 3*sizeof(float))*num_nodes + sizeof(ushort)*num_ins; //minimal amount of bytes needed to represent the tree 
			int r = size%sizeof(int4);
			size += r ? sizeof(int4)-r : 0;	//size of the tree padded to a multiple of sizeof(int4) (due to a GPU memory aligment requisite)
			return size;
		}
		__host__ void soalloc(int num_nodes, int num_ins)
		{
			void *ptr = malloc(getSize(num_nodes, num_ins));
			memset(ptr,0,getSize(num_nodes, num_ins));
			setOffs(num_nodes, ptr);
		}

		__host__ __device__ ushort	getParent	(int i) const {return parent[i].idx;}
		__host__ __device__ ushort	getSide		(int i) const {return parent[i].side;}
		__host__ __device__ ushort	getlChild	(int i) const {return lChild[i];}
		__host__ __device__ ushort	getrChild	(int i) const {return rChild[i];}
		__host__ __device__ ushort 	getInseq	(int i) const {return inseq[i];}
		__host__ __device__ float 	getBranch	(int i) const {return branch[i];}
		__host__ __device__ float	getdRoot	(int i) const {return dRoot[i];}
		__host__ __device__ float	getTrait	(int i) const {return vTraits[i];}

		__host__ __device__ void setParent	(ushort	val, int i)	{parent[i].idx	= val;}
		__host__ __device__ void setSide	(ushort	val, int i)	{parent[i].side	= val;}
		__host__ __device__ void setlChild	(ushort	val, int i)	{lChild[i]		= val;}
		__host__ __device__ void setrChild	(ushort	val, int i)	{rChild[i]		= val;}
		__host__ __device__ void setBranch	(float	val, int i)	{branch[i]		= val;}
		__host__ __device__ void setdRoot	(float	val, int i)	{dRoot[i]		= val;}
		__host__ __device__ void setInseq	(ushort	val, int i)	{inseq[i]		= val;}
		__host__ __device__ void setTrait	(float	val, int i)	{vTraits[i]		= val;}

};


void SoaTree::setOffs(int num_nodes, void* base) {
	parent 	= (paren_t*) base;
	lChild 	= (ushort*)	(parent+num_nodes);
	rChild 	= lChild	+ num_nodes;	
	branch 	=(float*)	(rChild+num_nodes);
	dRoot 	= branch	+ num_nodes;
	vTraits = dRoot +num_nodes;
	inseq 	=(ushort*)	(vTraits+num_nodes);
	
}

class HTree;

class DTree{

	protected:
		void *base;
		ushort nNodes;		// quantity of nodes on the tree(s) (including inserting species)
		ushort nInsSpc;		// quantity of absent species to be inserted
		ushort idxInsSpc;	// starting index for insertion of new species
		ushort idxInsAnc;	// starting index for insertion of new ancestors
		uint nTrees;		// quantity of trees holded by devData (default=1)
		size_t treeSize;	// size of one tree padded to multiple of sizeof(int4) (due to a GPU memory aligment requisite)
		ushort nLeafs;			// quantity of Leafs in the tree
		SoaTree devData;	// struct of arrays to hold trees' data
		ushort nElemMoran;   // number of elements in each tree that it's use to calculate the I-moran
		
	public:

		__host__ bool compareTo(HTree *h_tree);
		__host__ DTree() = default;
		__host__ DTree(int nNodes, int nInsSpc, int idxInsSpc, int idxInsAnc, int nTrees, size_t treeSize, void* ptr):
						nNodes(nNodes),
						nInsSpc(nInsSpc),
						idxInsSpc(idxInsSpc), 
						idxInsAnc(idxInsAnc),
						nTrees(nTrees),
						treeSize(treeSize){base=ptr; devData.setOffs(nNodes,ptr);}
		
		/* TODO: THROW OVER/UNDERFLOW EXCEPTION */
		__host__ __device__ ushort	getnNodes	()		const {return nNodes;}
		__host__ __device__ ushort	getnInsSpc	()		const {return nInsSpc;}
		__host__ __device__ ushort	getIdxInsSpc()		const {return idxInsSpc;}
		__host__ __device__ ushort	getIdxInsAnc()		const {return idxInsAnc;}
		__host__ __device__ uint	getnTrees	()		const {return nTrees;}
		__host__ __device__ size_t	getSize		()		const {return treeSize;};
		__host__ 			SoaTree getDevData	()		const {return devData;};
		__host__ __device__	ushort	getnLeafs	() 		const {return nLeafs;}
		__host__ __device__	ushort	getnElemMoran	() 	const {return nElemMoran;}
		

		__host__ __device__ ushort	getParent	(int i)	const {return devData.getParent(i);}
		__host__ __device__ ushort	getSide		(int i)	const {return devData.getSide(i);}
		__host__ __device__ ushort	getlChild	(int i)	const {return devData.getlChild(i);}
		__host__ __device__ ushort	getrChild	(int i)	const {return devData.getrChild(i);}
		__host__ __device__ float	getBranch	(int i)	const {return devData.getBranch(i);}
		__host__ __device__	float	getdRoot	(int i) const {return devData.getdRoot(i);}
		__host__ __device__ ushort	getInseq	(int i)	const {return devData.getInseq(i);}
		__host__ __device__	float	getTrait	(int i) const {return devData.getTrait(i);}


		__host__ __device__ void	setnElemMoran	(int val)	{nElemMoran = val;}
		__device__ void	setTreeIdx(int i){devData.setOffs(nNodes,base+treeSize*i);}

		/* TODO: THROW OVER/UNDERFLOW EXCEPTION */
		__device__ void	setParent	(ushort	val, int i)	{devData.setParent(val,i);}
		__device__ void	setSide		(ushort	val, int i)	{devData.setSide(val,i);}
		__device__ void	setlChild	(ushort	val, int i)	{devData.setlChild(val,i);}
		__device__ void	setrChild	(ushort	val, int i)	{devData.setrChild(val,i);}
		__device__ void	setBranch	(float	val, int i)	{devData.setBranch(val,i);}
		__device__ void	setdRoot	(float	val, int i)	{devData.setdRoot(val,i);}
		__device__ void	setInseq	(ushort	val, int i)	{devData.setInseq(val,i);}
		__device__ void	setTrait	(float	val, int i)	{devData.setTrait(val,i);}
	
		/* copy from GPU all the trees holded by the object and print them on the standard output  */
		__host__ void print(unordered_map<int,string> names);
		__host__ void print(unordered_map<int,string> names, int i);
		__host__ void free(){CHECK(hipFree(devData.getPtr()))}

		/* Calculate new Newick File */
		__host__ void toNewick(unordered_map<int,string> names);
		__host__ string calculateNewick(unordered_map<int,string> names,  SoaTree ht, int idRaiz);
		__host__ void newickToFile(string newick);

};


void DTree::toNewick(unordered_map<int,string> names){ 

	size_t rep_size = treeSize*nTrees;	
	void* h_replics = malloc(rep_size);
	CHECK(hipMemcpy(h_replics, devData.getPtr(), rep_size, hipMemcpyDeviceToHost));	
	SoaTree ht;
	int indexThree;
	string  newickFile="";
	for(indexThree=0; indexThree<nTrees; indexThree++){ // total of threes
		ht.setOffs(nNodes, h_replics+(treeSize*indexThree));// get the pointer set for a tree i
		// Ainda precisa resetar as variaveis str para cada nova interação
		newickFile += "#";
		newickFile += to_string(indexThree +1);
		newickFile += "\n";
		newickFile += calculateNewick(names, ht, nNodes -1);
		newickFile += "\n\n"; 
		// have save the newick in to a file
	}
	
	newickToFile(newickFile);
}

string DTree::calculateNewick(unordered_map<int,string> names, SoaTree ht, int idRaiz ){ 
	string  str_tmp,  str_float;

	//str_tmp = "";
   	//str_float = ""; // idRaiz = nNodes -1 , get the last element in the vector, that's the root
	if ( ht.getlChild(idRaiz)  == NOCHILD) { // left child of the root // Não tem filhos // nz_f1 = filho da esquerda do no
		if ((idRaiz) < 0 || (idRaiz) > (nNodes-1)) // num of nodes
			printf("ERRO %d\n", (idRaiz));
		else
			str_tmp += names[idRaiz]; 
		str_tmp += ":";
		str_float += to_string(ht.getBranch(idRaiz)); 
		str_tmp += str_float;
		return str_tmp;
	} else { // Has child 
		str_tmp +="(";
		str_tmp += calculateNewick(names, ht, ht.getlChild(idRaiz)); 
		str_tmp += ",";
		str_tmp +=  calculateNewick(names, ht, ht.getrChild(idRaiz)); 
		str_tmp += ")";
		str_tmp += names[idRaiz];

		if(nNodes -1 != idRaiz){ // if the element is not the root
			str_tmp +=":";
			str_float += to_string( ht.getBranch(idRaiz)); 
			str_tmp += str_float;
		}
		
		return str_tmp;
	}
	
}

void DTree::newickToFile(string newick ){ 
	
	ofstream ofFile;
	ofFile.open( "newNewick.tree" );
	ofFile<<newick;
	ofFile.close();

	/* case if needs to append to an existent newick file
	ifstream inFile;
	ofstream ofFile;
	string backup="", str="";

	inFile.open( "newNewick.tree" );

	if(!inFile){ // there's not the file in the folder

		ofFile.open( "newNewick.tree" ); // create a new file if there's not already a newick file
		ofFile<<newick;

	}

	else{ // there's a file and we gotta update it

		while (std::getline(inFile, str))
		{
		  backup += str;
		  backup.push_back('\n');
		} 

		backup += newick;
		backup.push_back('\n');

		ofFile.open("newNewick.tree"); // create a new file 
		ofFile<<backup;

		inFile.close();
	}

	ofFile.close();	
	*/
}

void DTree::print(unordered_map<int,string> names){

	size_t rep_size = treeSize*nTrees;	
	void* h_replics = malloc(rep_size);
	CHECK(hipMemcpy(h_replics, base, rep_size, hipMemcpyDeviceToHost));	
	SoaTree ht;
	string aux;
	int i,j;	
	cout.precision(4);
	cout.setf(ios::fixed, ios::floatfield);	
	cout << endl;
	for(i=0; i<nTrees; i++){
		cout<<"tree #"<<i<<endl;
		ht.setOffs(nNodes, h_replics+(treeSize*i));		
		for(j=0; j<nNodes; j++){
			aux = names[j]+"("+to_string(j)+")";
			cout << left << setw (40) << aux;
		} 
		cout << endl;
		for(j=0; j<nNodes; j++) {
			aux = ht.getParent(j)!=NOPARENT ? names[ht.getParent(j)]+"("+to_string(ht.getParent(j))+")" : "-1";
			cout << left << setw (40) << aux;
		} 
		cout << endl;
		for(j=0; j<nNodes; j++) {
			aux = ht.getSide(j)==1 ? "left" : "right";
			cout << left << setw (40) << aux;
		} 
		cout << endl;
		for(j=0; j<nNodes; j++) {
			aux = ht.getlChild(j)!=NOCHILD ? names[ht.getlChild(j)]+"("+to_string(ht.getlChild(j))+")" : "-2";
			cout << left << setw (40) << aux;	
		}
		cout << endl;
		for(j=0; j<nNodes; j++) {
			aux = ht.getrChild(j)!=NOCHILD ? names[ht.getrChild(j)]+"("+to_string(ht.getrChild(j))+")" : "-2";
			cout << left << setw (40) << aux;
		}
		cout << endl;
		for(j=0; j<nNodes; j++) cout << left << setw (40) << ht.getBranch(j); cout << endl;
		for(j=0; j<nNodes; j++) cout << left << setw (40) << ht.getdRoot(j); cout << endl;
		for(j=0; j<nInsSpc; j++)cout << names[ht.getInseq(j)] << "("<< ht.getInseq(j) <<") ";
		cout << endl << endl;
	}
}

void DTree::print(unordered_map<int,string> names, int i){

	size_t rep_size = treeSize*nTrees;
	void* h_replics = malloc(rep_size);
	CHECK(hipMemcpy(h_replics, devData.getPtr(), rep_size, hipMemcpyDeviceToHost));
	SoaTree ht;
	string aux;
	int j;
	cout.precision(4);
	cout.setf(ios::fixed, ios::floatfield);
	cout << endl;
	cout<<"tree #"<<i<<endl;
	ht.setOffs(nNodes, h_replics+(treeSize*i));
	for(j=0; j<nNodes; j++){
		aux = names[j]+"("+to_string(j)+")" + to_string(ht.getTrait(j));
		cout << left << setw (40) << aux ;
	}
	cout << endl;
	for(j=0; j<nNodes; j++) {
		aux = ht.getParent(j)!=NOPARENT ? names[ht.getParent(j)]+"("+to_string(ht.getParent(j))+")" : "-1";
		cout << left << setw (40) << aux;
	}
	cout << endl;
	for(j=0; j<nNodes; j++) {
		aux = ht.getSide(j)==1 ? "left" : "right";
		cout << left << setw (40) << aux;
	}
	cout << endl;
	for(j=0; j<nNodes; j++) {
		aux = ht.getlChild(j)!=NOCHILD ? names[ht.getlChild(j)]+"("+to_string(ht.getlChild(j))+")" : "-2";
		cout << left << setw (40) << aux;
	}
	cout << endl;
	for(j=0; j<nNodes; j++) {
		aux = ht.getrChild(j)!=NOCHILD ? names[ht.getrChild(j)]+"("+to_string(ht.getrChild(j))+")" : "-2";
		cout << left << setw (40) << aux;
	}
	cout << endl;
	for(j=0; j<nNodes; j++) cout << left << setw (40) << ht.getBranch(j); cout << endl;
	for(j=0; j<nNodes; j++) cout << left << setw (40) << ht.getdRoot(j); cout << endl;
	for(j=0; j<nInsSpc; j++)cout << names[ht.getInseq(j)] << "("<< ht.getInseq(j) <<") ";
	cout << endl << endl;

}

class HTree: public DTree{	
	private:
		SoaTree hostData;					// struct of arrays to hold the trees' data 		
		unordered_map<int, string> name;	// names of taxa fetched from newickf and PUT file
		ifstream newickf;					// stream object to manage input newick file
		ifstream putf;						// stream object to manage input PUT file
		ifstream traits;					// stream object to manage traits from the tree
		int devId;							// id of the GPU where lies the tree 
		__host__ void setParams(string &fileLine, vector<string> &filePut, vector<string> &fileTraits);
		__host__ void parseTree(string fileLine, vector<string> filePut);
		__host__ void parseTraits(vector<string> fileTraits);
	public:
		__host__ HTree() = default;
		__host__ HTree(int dev_id=0, string nw_fname = "newick.tree", string pt_fname="put.list", string tr_fname="traits.txt");
		__host__ DTree& gpuRep(int num_reps) const;
		
		/* TODO: THROW OVER/UNDERFLOW EXCEPTION */		
		__host__ void setParent (int 	val, int i)	{hostData.setParent(val,i);}
		__host__ void setSide	(int 	val, int i)	{hostData.setSide(val,i);}
		__host__ void setlChild (int 	val, int i)	{hostData.setlChild(val,i);}
		__host__ void setrChild (int 	val, int i)	{hostData.setrChild(val,i);}
		__host__ void setBranch (float 	val, int i)	{hostData.setBranch(val,i);}
		__host__ void setdRoot  (float 	val, int i) {hostData.setdRoot(val,i);}
		__host__ void setInseq 	(int 	val, int i) {hostData.setInseq(val,i);}
		__host__ void setName 	(string val, int i) {name[i]=val;}
		__host__ void setTrait  (float 	val, int i) {hostData.setTrait(val,i);}
		
		/* TODO: THROW OVER/UNDERFLOW EXCEPTION */
		__host__ ushort	getParent	(int i) const	{return hostData.getParent(i);}
		__host__ ushort	getSide		(int i) const	{return hostData.getSide(i);}
		__host__ ushort	getlChild	(int i) const	{return hostData.getlChild(i);}
		__host__ ushort	getrChild	(int i) const 	{return hostData.getrChild(i);}
		__host__ float	getBranch	(int i) const 	{return hostData.getBranch(i);}
		__host__ float	getdRoot	(int i) const 	{return hostData.getdRoot(i);}
		__host__ float	getTrait	(int i) const 	{return hostData.getTrait(i);}
		__host__ ushort	getInseq	(int i) const 	{return hostData.getInseq(i);}
		__host__ string	getName		(int i)			{return name[i];}
		__host__ unordered_map<int, string> getNames(){return name;}
};

HTree::HTree(int dev_id, string nw_fname, string pt_fname, string tr_fname){	
	long long start_time, end_time;
	struct timeval tv;

	void * d_tree;
	double time_spent;
	devId = dev_id;
	nTrees=1;
	CHECK(hipSetDevice(devId));

	START_TIMER();

	newickf.open(nw_fname);
	FERR(newickf);

	putf.open(pt_fname);
	FERR(putf);	

	traits.open(tr_fname);
	FERR(traits);

	string fileLine;
	vector<string> filePut;
	vector<string> fileTraits;
	setParams(fileLine,filePut,fileTraits);	

	hostData.soalloc(nNodes,nInsSpc);
	treeSize = hostData.getSize(nNodes,nInsSpc);
	parseTree(fileLine,filePut);
	parseTraits(fileTraits);
	newickf.close();
	putf.close();
	traits.close();
	STOP_TIMER(time_spent);
	cout<<"\ntotal time spent to parse the files: "<<time_spent<<"s\n";

	//make a copy of the tree on device side
	START_TIMER();
	CHECK(hipMalloc(&d_tree, treeSize));
	CHECK(hipMemcpy(d_tree, hostData.getPtr(), treeSize, hipMemcpyHostToDevice));	
	STOP_TIMER(time_spent);
	cout<<"\ntotal time spent to copy backbone tree to GPU: "<<time_spent<<"s\n";
	base=d_tree;
	devData.setOffs(nNodes, d_tree);
}


//compare argument tree to all the trees within object
bool DTree::compareTo(HTree *h_tree){
	if(treeSize != h_tree->getSize() || idxInsSpc != h_tree->getIdxInsSpc() || idxInsAnc != h_tree->getIdxInsAnc())
		return false;
	SoaTree tree;
	size_t rep_size = treeSize * nTrees;	
	void* h_replics = malloc(rep_size);
	CHECK(hipMemcpy(h_replics, devData.getPtr(), rep_size, hipMemcpyDeviceToHost));
	cout.precision(3);
	cout.setf(ios::fixed, ios::floatfield);	
	for(int j=0; j<nTrees; j++){
		tree.setOffs(nNodes, h_replics+treeSize*j);
		for(int i=0; i<nNodes; i++){
			if(	tree.getdRoot(i)	!= h_tree->getdRoot(i)	||
				tree.getBranch(i)	!= h_tree->getBranch(i) ||
				tree.getParent(i)	!= h_tree->getParent(i)	||
				tree.getSide(i)		!= h_tree->getSide(i)	||
				tree.getlChild(i)	!= h_tree->getlChild(i)	||
				tree.getrChild(i)	!= h_tree->getrChild(i)	)
					return false;
		}
	}
	return true;
}

void HTree::setParams(string &fileLine, vector<string> &filePut, vector<string> & fileTraits){	

	int fileLines=0;
	int aParen=0, fParen=0, comma=0;
	int quantElementosFile;
	char c;
	string currElement, aux;

	// read newick file
	while (newickf.get(c)) {
		fileLine +=c;
		fileLines++;

	}

	// read puts file
	nInsSpc = 0; // inicializar durante a construção
	aux = "";
	while (getline (putf,aux)) //enquanto end of file for false continua
    {      
      filePut.push_back(aux);                             
      nInsSpc++;
    }

    // read traits file
    aux = "";
	while (getline (traits,aux)) //enquanto end of file for false continua
    {      
      fileTraits.push_back(aux);                             
    }

	quantElementosFile = fileLines; // qnts elementos o arquivo tem	
	// primeira varredura apenas para verificar inconsistencias
	for(int i = 0; i < quantElementosFile; i++){ // faz uma varredura no arquivo
		currElement = fileLine[i];
		if(currElement == "(") aParen++;
		if(currElement == ")") fParen++;
		if(currElement == ",") comma++;
	}
	if(aParen != fParen){

		cout<< "Arquivo inconsistente, parentes não balanceados" <<endl;
		throw;
	}	

	nLeafs = aParen; // total of leafs
	idxInsSpc = nLeafs +1; // nos folhas
	nNodes = (aParen * 2) + (nInsSpc * 2) +2;
	idxInsAnc = nNodes - aParen -1;	
}


void HTree::parseTree(string fileLine, vector<string> filePut) {

	int posParent = -1;
	string leaf =" ", ancestral =" ", currElement=" ", parent = " "; // salva o atual e o ultimo elemento
	string leftChild=" ", rightChild= " ", comprimeRamoLeft ="", comprimeRamoRight = "";
	int auxiliarNumNos =0, auxiliarGeral =0, auxilarPreencherVetor =0; // usado para fazer as trocas de elementos no vetor
	int indexleftChild =-1, indexrightChild =-1;
	bool alphabeticModeOn = false; 
	
	// regex
	int quantFolhas = idxInsSpc;
	
	smatch m;
  	regex e ("\\([^()]+\\)");
  	regex folhas("\\([A-z0-9_+.#]+|,[A-z0-9_+.#]+"); // achar todas as folhas e separar no vetor
  	regex internos("\\)[A-z0-9_+.#]+|\\)[:;]");

  	// fill empty names
    size_t pos = 0;
    int n_unamed=1;
    while ((pos = fileLine.find("):", pos)) != std::string::npos) {
         fileLine.replace(pos, 2, ")#"+to_string(n_unamed)+":");
         pos += to_string(n_unamed).length()+3;
         n_unamed++;
    }
    // fill root's empty name (if absent)
    for(pos=fileLine.length(); fileLine[pos]!=';'; pos--);
    if(fileLine[--pos]==')')
    	fileLine.replace(pos,2,")#"+to_string(n_unamed)+";");
    // fill new ancestors' names
    for(int i=0; i<getnInsSpc();i++)
    	setName("na#"+to_string(i+1),getIdxInsAnc()-i); //the new ancestors' insertions order is backward oriented
  	//cout << "FileLine: " << fileLine << endl << endl;	
	for(int i=0;i<nNodes;i++){		
		setParent(NOPARENT,i);
		setlChild(NOCHILD,i);
		setrChild(NOCHILD,i);
		setBranch(0,i);
		setdRoot(0,i);		
	}
  	// preencher vetor com todas as species
	// usando o regex para pegar todos os quantFolhas	

	string copyNewick = fileLine;
	while (std::regex_search (copyNewick,m,folhas)) {
	    for (int i=0; i<m.size(); ++i) {
	    	auxiliarGeral = m.position(i)+1; // posicão do match (sem o '(' ou ',')
	    	leaf = copyNewick[auxiliarGeral++];	    	
	    	while(copyNewick[auxiliarGeral]!=':')
	    		leaf += copyNewick[auxiliarGeral++];	    	
  		}
		setName(leaf,auxilarPreencherVetor++);
	    copyNewick = m.suffix().str();
  	}
  	// preencher vetor com todas as species
	// usando o regex para pegar todos os nos internos	
	auxilarPreencherVetor = quantFolhas + (nInsSpc * 2) + 1;
	copyNewick = fileLine;
	while (std::regex_search (copyNewick,m,internos)) {
		ancestral = "";
	    for (int i=0; i<m.size(); ++i) {

	    	auxiliarGeral = m.position(i) +1; // posicão do match
		    while(copyNewick[auxiliarGeral]!=':' && copyNewick[auxiliarGeral]!=';') {
		    	ancestral += copyNewick[auxiliarGeral++];	    			
		    }		    
  		} 
  		setName(ancestral,auxilarPreencherVetor);
  		auxilarPreencherVetor++;
	    copyNewick = m.suffix().str();
  	}  	  	 
  	setParent(NOPARENT,nNodes-1); // no raiz não tem um pai
  	
	// logica se da no principio de achar todos os nos folhas pares, em cada loop, dai verificamos o seu devido pai
	// e os "eliminamos" da arvore, criando novos filhos folhas.
	// Para isso, estamos usando a biblioteca Redex, para achar os matchs e fazer o replace em seguida.
	// links: http://www.cplusplus.com/reference/regex/regex_search/
	//		  http://www.cplusplus.com/reference/regex/match_results/position/
	//        http://www.cplusplus.com/reference/regex/regex_replace/
	
	
	//regex logica
	// enquanto tivermos nos para buscar, vamos tirar as folhas
	// sobrara no final apenas o pai raiz
 
	int numTotalNos = nNodes-(2*nInsSpc)-1; 
	while(auxiliarNumNos < numTotalNos -1){	

		leftChild = "";
		rightChild = "";
		comprimeRamoLeft = "";
		comprimeRamoRight = "";
		std::regex_search ( fileLine, m, e );
    	
    	currElement = fileLine[m.position(0)]; // primeiro paranteses dos nos folhas achados    	
    	auxiliarGeral = m.position(0);

    	/* read everything until the ':' charactere is reached */
    	while(fileLine[++auxiliarGeral]!=':')
    		if(fileLine[auxiliarGeral]!=' ') leftChild += fileLine[auxiliarGeral];

   		while(fileLine[++auxiliarGeral]!=',')
    		if(fileLine[auxiliarGeral]!=' ') comprimeRamoLeft += fileLine[auxiliarGeral];

    	while(fileLine[++auxiliarGeral]!=':')
    		if(fileLine[auxiliarGeral]!=' ') rightChild += fileLine[auxiliarGeral];

   		while(fileLine[++auxiliarGeral]!=')')
    		if(fileLine[auxiliarGeral]!=' ') comprimeRamoRight += fileLine[auxiliarGeral];

	    auxiliarGeral++;

    	/* fetch name of the internal node (until ':') or of the root (until ';') */
    	parent="";
    	while(fileLine[auxiliarGeral]!=':' && fileLine[auxiliarGeral]!=';') {
	    	parent += fileLine[auxiliarGeral++];	    			
	    }	 

  		// achar o index entao dos filhos tirados e do pai
    	for(int i=0; i<nNodes; i++){
    		if(name[i] == parent){
    			posParent = i;
    			if( (indexleftChild != -1) and (indexrightChild != -1) ) break; // parar se ja achou indexes
    		}
    		else if(name[i]==rightChild){    			
    			indexrightChild = i;
    			if( (indexleftChild != -1) and (posParent != -1) ) break; 
    		}
    		else if(name[i]==leftChild){    			
    			indexleftChild = i;
    			if( (indexrightChild != -1) and (posParent != -1) ) break;
    		}
    	}
    	// preencher vetores
    	setParent(posParent,indexleftChild);
    	setSide(1,indexleftChild);
    	setParent(posParent,indexrightChild);
    	setSide(0,indexrightChild);
    	setlChild(indexleftChild,posParent);
    	setrChild(indexrightChild,posParent);
    	// comprimento do ramo
    	try{
	    	setBranch(atof(comprimeRamoRight.c_str()),indexrightChild);
	    	setBranch(atof(comprimeRamoLeft.c_str()),indexleftChild);
    	}catch(exception e){

    	}

	  	fileLine = m.prefix().str()+m.suffix().str();

	  	posParent = -1;
	  	// reset variaveis
  		rightChild = "";
  		leftChild = "";
  		comprimeRamoLeft = "";
  		comprimeRamoRight = "";
  		indexrightChild = -1;
  		indexleftChild = -1;
		auxiliarNumNos = auxiliarNumNos + 2; // ou seja, foi retirado 2 filhos
	}
	 // preencher novos put
 	string auxiliarPut[2], auxiliar, put;
  	for (int linePut = 0; linePut < nInsSpc; linePut++)
  	{  	
  		auxiliar = filePut[linePut];
  		put = ""; 
  		auxiliarGeral = 0;
  		alphabeticModeOn = false; 		
	    for (int elemenIndex = 0; elemenIndex < auxiliar.length(); elemenIndex++)
	    {	
	       if (isspace(auxiliar[elemenIndex]) and alphabeticModeOn) 
	       {
	       		auxiliarPut[auxiliarGeral++] = put;	       		
	           	put = "";
	           	alphabeticModeOn = false;
	       }else{
	       		if ( !isspace(auxiliar[elemenIndex]) ){ 
	       			alphabeticModeOn = true;
	        		put += auxiliar[elemenIndex];
	        	}	        	
	       }	 
	    }
	    if(put != ""){
	    	auxiliarPut[auxiliarGeral] = put;
	    }
	    //insert no array especies
	    setName(auxiliarPut[0],quantFolhas+linePut);
	    for (int index = 0; index < nNodes; index++)
	    {
	    	if(name[index] == auxiliarPut[1]){
	    		if(index>=getIdxInsSpc())
	    			setParent(index,quantFolhas+linePut);
	    		else // if the MDCC is a leaf, make its parent become the new MDCC 
	    			setParent(getParent(index),quantFolhas+linePut);
	    		break;
	    	}
	    }	
  	} 
	// Calcular comprimento do ramo ate a raiz
	// usando busca em profundidade
	bool folhaDone = false;
	int visited=0;
	setBranch(0,nNodes-1);	//root has no branch
	setdRoot(0,nNodes-1); 	//root has no distance to himself
	int posRamo = getrChild(nNodes-1);//start with the root's right child;
	while(visited<quantFolhas*2-2){
		// primeiramente, faz uma busca profunda, pela esquerda(mas na vdd tanto faz), e busca um no leaf
		// com isso, sabemos a profundidade de todos os outros folhas, restando então apenas os nos internos
		// essa regra se aplica apenas para arvores filogeneticas
		while(not folhaDone){
			setdRoot(getdRoot(getParent(posRamo))+getBranch(posRamo),posRamo);
			if(getrChild(posRamo) == NOCHILD){ // ou seja, não tem filho(leaf)
				folhaDone = true;
				// temos então o comprimento de todos os folhas da arvore
				// atualizar de todas as folhas então				
				for (int i = 0; i < quantFolhas + nInsSpc; i++)
				{
					setdRoot(getdRoot(posRamo),i);
				}
				visited+= quantFolhas;
				posRamo = getParent(posRamo); // volta entao a posição ramo 1 posição, pois chegou no limite da arvore(leaf)				
				break;
			}
			visited++;
			posRamo = getrChild(posRamo); // proximo filho a direita
		}		
		// fazer a busca em profundidade agr para os nos internos
		// se os dois filhos da raiz, ja tiverem seus comprimentos achados,
		// entao significa q a busca em profundidade foi concluida

		// cheka se elemento atual ainda tem filho 
		if(getrChild(posRamo)!=NOCHILD){
			// se tiver filho da direita e o comp dele ainda n foi calculado
			if(getdRoot(getrChild(posRamo))==0){
				// nova posRamo é entao aquele filho da direita
				visited++;
				posRamo = getrChild(posRamo);
				setdRoot(getdRoot(getParent(posRamo))+getBranch(posRamo),posRamo);
			}
			 // se tiver filho da esquerda e o comp dele ainda n foi calculado
			else if(getdRoot(getlChild(posRamo))==0){
				// nova posRamo é entao aquele filho da direita
				visited++;
				posRamo = getlChild(posRamo);
				setdRoot(getdRoot(getParent(posRamo))+getBranch(posRamo),posRamo);
			}
			// ou seja, aquela sub arvore esta concluida
			else{				
				posRamo = getParent(posRamo); // volta entao a posição ramo 1 posição, pois chegou no limite da arvore(leaf)	
			}	
		}
	}
	//setup insertion sequence
	for(int i=0; i<getnInsSpc();i++) // insert new puts nequence
		setInseq(getIdxInsSpc()+i,i); // getIdxInsSpc return the first indice of the put to be insered
}

void HTree::parseTraits(vector<string> fileTraits) {

	int numElements =0, auxiliarGeral=0;
	string auxiliarTrait[2], auxiliar, trait;
	bool alphabeticModeOn = false; 
	for(string elem: fileTraits){

		if (numElements++ !=0) // first element is not consider, because it's a description of the file(ex: SppName	BodySize)
		{
			auxiliar = elem;
	  		trait = ""; 
	  		auxiliarGeral = 0;
	  		alphabeticModeOn = false; 		
		    for (int elemenIndex = 0; elemenIndex < auxiliar.length(); elemenIndex++)
		    {	
		       if (isspace(auxiliar[elemenIndex]) and alphabeticModeOn) 
		       {
		       		auxiliarTrait[auxiliarGeral++] = trait;	      		
		           	trait = "";
		           	alphabeticModeOn = false;
		           	
		       }else{
		       		if ( !isspace(auxiliar[elemenIndex]) ){ 
		       			alphabeticModeOn = true;
		        		trait += auxiliar[elemenIndex];
		        	}	        	
		       }	 
		    }
		    if(trait != ""){
		    	auxiliarTrait[auxiliarGeral] = trait; 
		    }

		    // calculate then, where in the vector each caracteristic value must be, by comparing where the element
		    // is in the vector of elements  

		    for (int index = 0; index < nNodes; index++)
		    {
		    	if(auxiliarTrait[0] == name[index]){ 
		    		setTrait(::atof(auxiliarTrait[1].c_str()),index);
		    		break;
		    	}
		    }

		  


		}
	}

	// cout<<"Original Trais Inf\nName\t\tTrait"<<endl;
	// for (int index = 0; index < nNodes; index++)
	// 	{	
	// 	    cout<<getName(index)<<"\t\t" <<getTrait(index)<<endl;
		    	
	// 	}
		
}

//creates 'num_reps' replics of the tree holded by the object, inside GPU Global memory, and return a reference to them
DTree& HTree::gpuRep(int num_reps) const{	
	size_t rep_size = treeSize * num_reps;
	void *d_replics;
	CHECK(hipMalloc(&d_replics, rep_size));
	hipDeviceProp_t device;
	CHECK(hipGetDeviceProperties(&device,devId));
	int threads = device.warpSize*16;	//TODO: FIGURE OUT WHICH MULTIPLE IS THE BEST
	int blocks = (rep_size/sizeof(int4) + (threads-1)) / threads;
	dim3 grid = dim3(blocks);
	dim3 block = dim3(threads);
	modcpy<<<grid, block>>>(d_replics,devData.getPtr(),rep_size,treeSize);
	CHECK(hipDeviceSynchronize());
	return *new DTree(nNodes,nInsSpc,idxInsSpc,idxInsAnc,num_reps,treeSize,d_replics);
}

//create all necessary seeds to massive GPU randomize
__global__ void setup_kernel(long long seed, hiprandState_t* devStates, ushort N){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int i;
    for(i=idx;i<N;i+=gridDim.x*blockDim.x)
    	hiprand_init(seed, i, 0, &devStates[i]);
}

//trees' exapansions
__global__ void insertion(DTree tree, hiprandState_t* devStates){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	hiprandState state;
	unsigned int i,j,t;
	int taxon, mdcc;
	int ancidx; 	//the put's parent node created to represent the cladogenesis
	int grandpa;
	unsigned int k;

	float depth;	//depth in which the put will be inserted down the subtree rooted at mdcc
	float height;	//height of the tree (distance from leaf to root)
	
	for(k=idx;k<tree.getnTrees();k+=gridDim.x*blockDim.x){	
		tree.setTreeIdx(k); // Serva para pegar qual arvore especifica vc quer mecher, Dtree é um vetor de tree
	    state = devStates[k];
	    height = tree.getdRoot(0); //height of the tree (distance from leaf to root)

	    if (tree.getnInsSpc() > 1) {
		for (i=0; i<tree.getnInsSpc()-1; i++) { // Serva para embaralhar cada replica de arvore criada
			j = i + hiprand(&state) / (UINT_MAX/(tree.getnInsSpc()-i)+1);
			t = tree.getInseq(j);

			tree.setInseq(tree.getInseq(i),j); //(Value, Position) apos embaralhar, cada um tera um novo vetor de sequencia
			tree.setInseq(t,i); // used these 2 lines above to also switch the values of the traits vector

	    }
	    
	    float sum;
	    ushort put; //current put going to be inserted
		for(i=0; i<tree.getnInsSpc(); i++){
			t = hiprand(&state);	//path
			put = tree.getInseq(i);
			mdcc = tree.getParent(put);	
			depth = hiprand_uniform(&state) * (height-tree.getdRoot(mdcc));
			taxon = mdcc;
			sum=0;
			do{		
				t>>=1;
				taxon = t&1 ? tree.getlChild(taxon) : tree.getrChild(taxon);
				sum+= tree.getBranch(taxon);			
			}while(sum<depth);
			//after the loop, taxon is the sister clade
			grandpa = tree.getParent(taxon);
			ancidx = tree.getIdxInsAnc()-(put-tree.getIdxInsSpc());	//calculate corresponding ancestor node		
			if(t&1){	//if came from the left
				tree.setrChild(put,ancidx);		//put become the right child
				tree.setlChild(taxon,ancidx);	//the sister clade continue being at left
				tree.setlChild(ancidx,grandpa);//the put's parent node takes place of the sister's clade side
			}			
			else{	//if came from the right
				tree.setlChild(put,ancidx);		//put become the left child
				tree.setrChild(taxon,ancidx);	//the sister clade continue being at right
				tree.setrChild(ancidx,grandpa);//the put's parent node takes place of the sister's clade side
			}
			tree.setParent(grandpa,ancidx);				//set up new ancestor's parent (same of the sister group)
			tree.setSide(t&1,ancidx);									//set up new ancestor's side (same of the sister group)
			tree.setParent(ancidx,put);									//set up PUT's parent
			tree.setSide(!(t&1),put);									//set up PUT's side (the sister's reverse)
			tree.setParent(ancidx,taxon);								//set up sister's new parent
			tree.setBranch(tree.getBranch(taxon)-(sum-depth),ancidx);	//set up new ancestor's branch
			tree.setBranch(sum-depth,taxon);							//set up sister's new branch length
			tree.setBranch(height-(tree.getdRoot(mdcc)+depth),put);		//set up PUT's branch length
			tree.setdRoot (tree.getdRoot(grandpa)+tree.getBranch(ancidx),ancidx);	//set up new ancestor's distance to the root
		}	
	}
}

}

__host__ __device__ int row_index( int i, int M ){ // retorna o indice da linha
	M--;
    float m = M;
    float row = (-2*m - 1 + sqrt( (4*m*(m+1) - 8*(float)i - 7) )) / -2;
    if( row == (float)(int) row ) row -= 1;
    return (int) row;
}

__host__ __device__ int column_index( int i, int M ){ // retorna o indice da coluna
    int row = row_index( i, M);
    M--;
    return 1 + (i - M * row + row*(row+1) / 2);
}

//generate the patristic distance matrixes to all the replics
__global__ void patrix(DTree tree, float* d_matrix){

		tree.setTreeIdx(blockIdx.x);
		uint idx = threadIdx.x;
		ushort row, col, taxon;
		unsigned long long row_bmp, col_bmp; 
		ushort row_len, col_len;
		ushort N = tree.getnNodes();
		ushort nleafs = (N+1)/2;
		uint msize = nleafs*(nleafs-1)/2;

		extern __shared__ ushort s[];

		ushort *parent = s;
		ushort *lchild = parent+N;
		ushort *rchild = lchild+N;

		uint i;

		//separated loops to favor coalesced access
		for(i=idx;i<N;i+=blockDim.x)
				parent[i] = tree.getParent(i);
		for(i=idx;i<N;i+=blockDim.x)
				lchild[i] = tree.getlChild(i);
		for(i=idx;i<N;i+=blockDim.x)
				rchild[i] = tree.getrChild(i);

		__syncthreads();

		for(i=idx;i<msize;i+=blockDim.x)
		{
			row=row_index(i,nleafs);
			col=column_index(i,nleafs);
			row_bmp=0;
			col_bmp=0;
			row_len=0;
			col_len=0;
			for(taxon=row; parent[taxon]!=NOPARENT; taxon=parent[taxon]){
				row_len++;
				row_bmp<<=1;
				row_bmp|=tree.getSide(taxon);
			}
			for(taxon=col; parent[taxon]!=NOPARENT; taxon=parent[taxon]){
				col_len++;
				col_bmp<<=1;
				col_bmp|=tree.getSide(taxon);
			}
			taxon=tree.getnNodes()-1; 	//start with the root
			if((row_bmp&1)==(col_bmp&1)){	//if the LCA isn't the root				
				do{
					taxon = row_bmp&1 ? lchild[taxon] : rchild[taxon]; // either row_bmp or col_bmp (same)
				 	row_bmp>>=1;
				 	col_bmp>>=1;
				 }while((row_bmp&1)==(col_bmp&1));
			}
			d_matrix[blockIdx.x*msize+i] = 2*(tree.getdRoot(row)-tree.getdRoot(taxon));
	}
}

__global__ void I_moran(DTree tree, float* d_matrix, int nrClass, float* nz_class_range, float* nz_class_value, float MeanY, float Variance){

	tree.setTreeIdx(blockIdx.x);
	uint idxThread = threadIdx.x; // Thread identifier 
	uint idxBlock = blockIdx.x;	  // Block identifier
	ushort row, col, indexI, indexJ;
	ushort nthreads, nLeafs, numElemThread;
 	float SumProdCross, SumW, w;
  	ushort symmetry, numElemMoran, base;

	__shared__ float sumTotal, sumTotalProdCross;
	extern __shared__ float classRange[];


	for(indexI=0;indexI<nrClass;indexI++){
		classRange[indexI] = nz_class_range[indexI];
	}

	nLeafs = (tree.getnNodes()+1)/2;
	numElemMoran = tree.getnElemMoran(); 

	base = idxBlock * numElemMoran;  // acessar od dados corretamente de cada arvore em seu devido bloco

	SumW = 0;

	if ( (nLeafs % 2) == 0) {
		nthreads = nLeafs - 1; // nLeafs é par: cada thread calcula nLeafs/2 distancias 
		numElemThread = nLeafs / 2; // quantidade de elementos por thread
	} else {
		nthreads = nLeafs;	 // nLeafs é ímpar: cada thread calcula (nLeafs-1)/2 distancias
		numElemThread = (nLeafs - 1) / 2;  // quantidade de elementos por thread
	}

	w = 1;
    symmetry = 2; //Symetric
	//Inicializa variaveis compartilhadas
	sumTotalProdCross = 0;
	sumTotal = 0;
	__syncthreads();//aguarda inicializacao das variaveis para continuar execução
	if (idxThread < nthreads) {
  		for(indexI=0;indexI<nrClass;indexI++){
    			SumProdCross = 0;
	    		SumW = 0;
			for( indexJ = idxThread*numElemThread; indexJ < numElemThread+(idxThread*numElemThread); indexJ++) {
				if (d_matrix[indexJ+base] > classRange[indexI] ) {

					if ( (indexI < nrClass-1) && (d_matrix[indexJ+base] <= classRange[indexI+1]) ) 
					{
						row = row_index(indexJ, nLeafs); 
						col = column_index(indexJ, nLeafs);
						SumW += (w*symmetry);
					
						SumProdCross += (((tree.getTrait(row) - MeanY) * (tree.getTrait(col) - MeanY))*symmetry);
					}
					else if(indexI >= nrClass-1){
						row = row_index(indexJ, nLeafs); 
						col = column_index(indexJ, nLeafs);
						SumW += (w*symmetry);
					
						SumProdCross += (((tree.getTrait(row) - MeanY) * (tree.getTrait(col) - MeanY))*symmetry);

					}
				
					
				}
			} 

			//Utilizar operacao atomica to make sure there's not concurrency for the value
			/*
				reads the 32-bit or 64-bit word old located at the address address in global or shared memory, 
				computes (old + val), and stores the result back to memory at the same address. These three operations 
				are  performed in one atomic transaction. The function returns old.
			*/
			atomicAdd(&sumTotalProdCross, SumProdCross);
			atomicAdd(&sumTotal, SumW);
			__syncthreads(); // espera todas as threads chegarem até aqui, pois foi feito uma operação atomica

			//apenas uma thread calcula o I de Moran
			if (threadIdx.x == 0){ 
				nz_class_value[(idxBlock*nrClass)+indexI] =  (nLeafs / sumTotal) * (sumTotalProdCross / Variance);     // I de Moran
				
				sumTotalProdCross = 0;
				sumTotal = 0;

			}
			__syncthreads(); // espera todas as threads chegarem até aqui
   		}
	}

}

float *nz_class_range; //Faixa para as classe de distância
float *nz_class_value; //Valores de I de Moran por classe de distância
float *nz_class_media; //Valores medios de I de Moran por classe de distância
float *nz_class_variance; //Variancia para cada classe de distância

/* Pointers to GPU / I-moran*/
float *nz_class_range_d; //Faixa para as classe de distância
float *nz_class_value_d; //Valores de I de Moran por classe de distância

int main(int argc, char *argv[]){	

	if(argc < 2 || argc >4){
		cout << "Usage: " << argv[0] << " #replications [newick putlist]" << endl;
		exit(EXIT_FAILURE);
	}
	
	long long start_time, end_time;
	struct timeval tv;

	int gpu=0; 
	double time_spent;
	int num_reps = atoi(argv[1]);	
	int index =0;
	HTree *tree = argc>2 ? new HTree(gpu,argv[2],argv[3]) : new HTree(gpu);
	
	CHECK(hipSetDevice(gpu));
	START_TIMER();
	DTree replics = tree->gpuRep(num_reps);
	STOP_TIMER(time_spent);
	cout<<"\ntotal time spent to replicate trees: "<<time_spent<<"s\n";

	cout << "nNodes: " << tree->getnNodes() << endl;
	cout << "nInsSpc: " << tree->getnInsSpc() << endl;
	cout << "idxInsSpc: " << tree->getIdxInsSpc() << endl;
	cout << "idxInsAnc: " << tree->getIdxInsAnc() << endl << endl;

/*
	if(replics.compareTo(tree))
		cout << "Data does match!" << endl;
	else
		cout << "Data doesn't match" << endl;
		
*/
	hiprandState_t *devStates;
	hipDeviceProp_t device;
	CHECK(hipGetDeviceProperties(&device,gpu));
	
	int threads = device.warpSize*16; //threads per block; TODO: FIGURE OUT WHICH MULTIPLE IS THE BEST
	int blocks = (num_reps + (threads-1)) / threads;
	dim3 grid(blocks), block(threads);
	
	START_TIMER();
	CHECK(hipMalloc((void**)&devStates, sizeof(hiprandState_t)*num_reps));	
	setup_kernel<<<grid,block>>>(1,devStates,num_reps);
	CHECK(hipDeviceSynchronize());
	

	insertion<<<grid,block>>>(replics,devStates);	// Nesse metodo é feito o embaralhamento do vetor de indices de cada tree
	

	CHECK(hipDeviceSynchronize());
	STOP_TIMER(time_spent);
	cout<<"\ntotal time spent to expand trees: "<<time_spent<<"s\n";

	//replics.print(tree->getNames(),0);
	//replics.toNewick(tree->getNames());

	START_TIMER();
	ushort nleafs = (replics.getnNodes()+1)/2;
	uint msize = nleafs*(nleafs-1)/2;
	

	float *d_matrix;
	CHECK(hipMalloc((void**)&d_matrix, sizeof(float)*msize*num_reps));
	patrix<<<num_reps,256,replics.getnNodes()*(sizeof(ushort)*3)>>>(replics, d_matrix);
	CHECK(hipDeviceSynchronize());
	STOP_TIMER(time_spent);
	cout<<"\ntotal time spent to generate patrixes: "<<time_spent<<"s\n";

	
	START_TIMER();
	float *h_matrix = (float*)malloc(sizeof(float)*msize*num_reps);
	CHECK(hipMemcpy(h_matrix, d_matrix, sizeof(float)*msize*num_reps, hipMemcpyDeviceToHost));
	STOP_TIMER(time_spent);
	cout<<"\ntotal time spent to copy patrixes back to CPU: "<<time_spent<<"s\n";

	/*****************************************************
	
	 	   C A L C U L A R   I   D E   M O R A N
	
	******************************************************/

	//Aloca posicoes em memoria para armazenar as classes de distância
	int nrClass = 4; // number of class ranges to be used(4 = padrao)
	float maiorDistancia=0, menorDistancia= h_matrix[0];
	float saltoVariancia; // usado para calcular a media entre maior e menor valor de distancia 
	float *d_mtx_lfs, *h_mtx_lfs; // save the matrix distance from the leafs
	CHECK(hipMalloc((void**)&d_mtx_lfs, sizeof(float)*msize*num_reps));
	h_mtx_lfs = (float*)malloc(sizeof(float)*msize*num_reps);

	nz_class_range = (float *) malloc((nrClass+1) * sizeof(float));
	nz_class_value = (float *) malloc((nrClass*num_reps) * sizeof(float));
	nz_class_media = (float *) malloc(nrClass * sizeof(float) );
	nz_class_variance = (float *) malloc(nrClass * sizeof(float) );

	int nElemMoran; // how many elements in each tree to calculate I-moran
	// Classes are difined in a equal way. Between largest and smallest value
	for (index=0;index<num_reps;index++){ // for to go through each matrix

		int countLeafs =  ((replics.getnNodes()/2) -1) /2; // how many leafs in that especific row
		int currentRow = 0; // control in which row it's
		float element = 0;
		int indexLefsMtx = 0;
		nElemMoran =0;
		for (int i = 0; i < msize; ++i) // for to go through each element of each matrix
		{
			element = h_matrix[index* msize + i];
			nElemMoran++;

			if (maiorDistancia < element)
				maiorDistancia = element;	
			if (menorDistancia > element)
				menorDistancia = element;

			// pass the values of the leaf's matrix distance to the vector  h_mtx_lfs
			h_mtx_lfs[index*sizeof(float)*nleafs + indexLefsMtx] = element;
			indexLefsMtx++;
			countLeafs--;
			if (countLeafs <1) // ead of leafs at that row
			{
				currentRow++; // go to next row, because there's no more leafs at that row
				countLeafs = (((replics.getnNodes()/2) -1) /2 ) - currentRow; // it's 1 element less for the next each row
				if(countLeafs <= 0){// there's no more leafs at this matrix 
					break; // come back to the extern for and start it again, for the next matrix
				}
				i +=(nleafs/2)+1; // everytime a row is done, it jumps total os leafs/2, to go to the begining
									// of the next row, where the leafs are
			}			

		}
		
	}

	replics.setnElemMoran(nElemMoran);


	saltoVariancia = (maiorDistancia - menorDistancia)/nrClass; // saltoVariancia é a "media" entre maior e menor, representando o saltoVariancia entre as classes
	for(index=0;index<nrClass;index++){
		nz_class_range[index] = menorDistancia;
		nz_class_value[index] = 0.0;
		menorDistancia += saltoVariancia;
	}
	nz_class_range[0] -= nz_class_range[0]/2; //para incluir distancias iguais ao menor valor
	nz_class_range[index] = maiorDistancia;


	CHECK(hipMalloc((void**)&nz_class_range_d, sizeof(float)*(nrClass+1)));  //+1 para guardar a faixa final da classe
	CHECK(hipMalloc((void**)&nz_class_value_d, sizeof(float)*nrClass*num_reps));

	/*Copiar dados para GPU*/
	CHECK(hipMemcpy(nz_class_range_d, nz_class_range, sizeof(float) * (nrClass+1), hipMemcpyHostToDevice));
	CHECK(hipMemcpy(nz_class_value_d, nz_class_value, sizeof(float) * nrClass * num_reps, hipMemcpyHostToDevice));
	CHECK(hipMemcpy(d_mtx_lfs, h_mtx_lfs, sizeof(float)*msize*num_reps, hipMemcpyHostToDevice));


	/* Data used to calculate I-moran
	   It's "cheaper" to calculate on host side
	*/ 
	size_t rep_size = replics.getSize()*replics.getnTrees();	
	void* h_replics = malloc(rep_size);
	CHECK(hipMemcpy(h_replics, replics.getDevData().getPtr(), rep_size, hipMemcpyDeviceToHost));	
	SoaTree ht;
	ht.setOffs(replics.getnNodes(), h_replics+(replics.getSize()*0));
	
	float variance, MeanY, SumW;

	SumW = 0;
  	variance = 0;
  	for (index=0;index<nleafs;index++){
    		SumW += ht.getTrait(index);
    		variance += + pow(ht.getTrait(index),2);
	}
  	MeanY = SumW / nleafs;
	variance -= (pow(SumW, 2) / nleafs);
	

	CHECK(hipGetDeviceProperties(&device,gpu));

	// Chamada I-moran
	CHECK(hipDeviceSynchronize());
	// int threadsMoran = device.warpSize*16; 
	// int blocksMoran = (replics.getnLeafs + (threadsMoran-1)) / threadsMoran;
	// dim3 gridMoran(blocksMoran), blockMoran(threadsMoran);
	// I_moran<<<gridMoran,blockMoran>>>();

	// testar 1,1 - sequencial(Ter certeza que esta funcionando)
	//I_moran<<<1,1>>>(replics, d_matrix, nrClass, nz_class_range_d, nz_class_value_d, MeanY, variance); 


	CHECK(hipDeviceSynchronize());
	START_TIMER();
	int aux = sizeof(float)*(nrClass+1);
	I_moran<<<num_reps,nleafs,aux>>>(replics, d_mtx_lfs, nrClass, nz_class_range_d, nz_class_value_d, MeanY, variance); 
	

	CHECK(hipDeviceSynchronize());
	STOP_TIMER(time_spent);
	cout<<"\ntotal time spent to calculate I_moran: "<<time_spent<<"s\n";
	
	//Traz os resultados de volta (GPU para Host), as medias são armazenadas no início do vetor
	CHECK(hipMemcpy(nz_class_value, nz_class_value_d, nrClass * sizeof(float) * num_reps, hipMemcpyDeviceToHost));

	//Calcula a media por classe e a variancia
	float media;
	int nrArvore;
	for(int i=0;i<nrClass;i++){
		media = 0;
		for (nrArvore=i;nrArvore<(num_reps*nrClass);nrArvore+=nrClass){
			media += nz_class_value[nrArvore];
		}
		nz_class_media[i] = media / num_reps;
	}
	//calculo da variancia	
	for(int i=0;i<nrClass;i++){
		media = 0;
		for (nrArvore=i;nrArvore<(num_reps*nrClass);nrArvore+=nrClass){
			media += pow((nz_class_value[nrArvore] -  nz_class_media[i]), 2);
		}
		nz_class_variance[i] = media / num_reps;
	}

	cout<<"\n\n\t############## I-Moran Results ##############\n"<<endl;
	cout<<"Numero de classes: "<<nrClass<<endl<<endl;
	for(int i=0;i<(nrClass);i++){

		cout<<"\tClasse "<<i+1<<":"<<endl;
		cout<<"Value: "<<nz_class_value[i]<<"\nMedia: "<<nz_class_media[i]<<"\nVariance: "<<nz_class_variance[i]<<endl<<endl;

	}


	CHECK(hipDeviceReset());	
	exit(EXIT_SUCCESS);	
}













